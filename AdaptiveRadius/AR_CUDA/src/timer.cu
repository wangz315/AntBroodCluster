/* Some simple convenience functions to make timing CUDA events a little less verbose.
 */

#include "timer.h"

Timer create_timer()
{
    Timer timer;
    hipEventCreate(&(timer.start));
    hipEventCreate(&(timer.stop));

    return timer;
}

void destroy_timer(Timer *timer)
{
    hipEventDestroy(timer->start);
    hipEventDestroy(timer->stop);
}

void start_timer(Timer *timer)
{
    hipEventRecord(timer->start);
}

void stop_timer(Timer *timer)
{
    hipEventRecord(timer->stop);
}

// Returns the elapsed time between the given events, in milliseconds
float get_time(Timer *timer)
{
    float millisec = 0;
    hipEventElapsedTime(&millisec, timer->start, timer->stop);
    
    return millisec;
}
