#include "hip/hip_runtime.h"
#include "kernels.h"


#define IA 16807
#define IM 2147483647
#define AM (1.0/IM)
#define IQ 127773
#define IR 2836
#define MASK 123459876


// prototypes
__device__ double atomicAddDouble(double* address, double val);
__device__ double randCuda(long *idum);
__device__ void move(int* clocks, ant_t* antArray, int dim);
__device__ double computeProbabilityOriginal(Lock* locks, int objectI, int x, int y, int radius, int dim, int numObjects, int* objectGrid, double* objectDist);
__device__ double computeProbability(Lock* locks, int objectI, int x, int y, int radius, int dim, int numObjects, int* objectGrid, double* objectDist);
__device__ void antRunDevOriginal(Lock* locks, int* antGrid, int* objectGrid, ant_t* antArray, object_t* objectArray, double* objectDist, int dim, int numAnts, int numObjects, int radius);
__device__ void antRunDev(Lock* locks, int* antGrid, int* objectGrid, ant_t* antArray, object_t* objectArray, double* objectDist, int dim, int numAnts, int numObjects, int radius);


// functions
__device__ double atomicAddDouble(double* address, double val)
{
	unsigned long long int* address_as_ull = 
	    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__device__ double randCuda(long *idum)
{
	int k;
	double ans;

	k =(*idum)/IQ;
	*idum = IA * (*idum - k * IQ) - IR * k;
	if (*idum < 0 ) *idum += IM;
	ans = AM * (*idum);
	return ans;
}

__device__ void move(int* clocks, Lock* locks, int* antGrid, ant_t* antArray, int dim)
{
	
	int factor = 1;
	int randNum = randCuda(&antArray[blockIdx.x].seed) * ((factor*2+1)*(factor*2+1)-1);// *(max-1), to avoid move to same location

	//increase 2nd half to avoid move to same location
	if(randNum >= ((factor*2+1)*(factor*2+1)/2))
	{
		randNum++;
	}

	int x = randNum % (factor*2+1) - factor + antArray[blockIdx.x].x;
	int y = randNum / (factor*2+1) - factor + antArray[blockIdx.x].y;

	x %= dim;
	y %= dim;

	if(x < 0)
	{
		x += dim;
	}
	
	if(y < 0)
	{
		y += dim;
	}	
		
	locks[x*dim+y].lock();
	if(antGrid[x*dim+y] == -1)
	{
		antGrid[antArray[blockIdx.x].x*dim+antArray[blockIdx.x].y] = -1;
		antArray[blockIdx.x].x = x;
		antArray[blockIdx.x].y = y;
		antGrid[x*dim+y] = blockIdx.x;
	}
	locks[x*dim+y].unlock();

}


__device__ double computeProbability(Lock* locks, int objectI, int x, int y, int radius, int dim, int numObjects, int* objectGrid, double* objectDist)
{
	int objectJ;
	int xi, yj;
	int xs = x - radius;
	int ys = y - radius;
	double sim = 0.0;
	double total = 0.0;

	for(int i = 0; i < radius*2+1; i++)
	{
		xi = (xs + i) % dim;

		for(int j = 0; j < radius*2+1; j++)
		{
			if (j != x && i != y)
			{
				yj = (ys + j) % dim;

				if(xi < 0)
				{
					xi += dim;
				}

				if(yj < 0)
				{
					yj += dim;
				}

				objectJ = objectGrid[xi*dim+yj];
				if(objectJ > -1)
				{
					sim = objectDist[objectI*numObjects+objectJ];
					total += exp(-1*sim);
				}
			}
		}
	}

	int area = (powf((radius*2) + 1, 2) - 1);
	double density = total / area;
	density = max(min(density, 1.0), 0.0);
	double temp = exp(-1 * powf(density, 2));
	return  (1 - temp) / (1 + temp);
}


__device__ void antRunDev(int* clocks, Lock* locks, int* antGrid, int* objectGrid, ant_t* antArray, object_t* objectArray, double* objectDist, int dim, int numAnts, int numObjects, int radius)
{
	int x = antArray[blockIdx.x].x;
	int y = antArray[blockIdx.x].y;
  

	if(objectGrid[x*dim+y] > -1)
	{
		if(antArray[blockIdx.x].objectId == -1)
		{
			double Ppick = 1 - computeProbability(locks, antArray[blockIdx.x].objectId, x, y, radius, dim, numObjects, objectGrid, objectDist);


			if(Ppick > randCuda(&antArray[blockIdx.x].seed))
			{
				antArray[blockIdx.x].objectId = objectGrid[x*dim+y];
				objectGrid[x*dim+y] = -1;

			}
			else
			{
				move(clocks, locks, antGrid, antArray, dim);
			}
		}
		else
		{
			move(clocks, locks, antGrid, antArray, dim);
		}
	}
	else
	{
		if(antArray[blockIdx.x].objectId > -1)
		{
			double Pdrop = computeProbability(locks, antArray[blockIdx.x].objectId, x, y, radius, dim, numObjects, objectGrid, objectDist);

			if(Pdrop > randCuda(&antArray[blockIdx.x].seed))
			{
				objectGrid[x*dim+y] = antArray[blockIdx.x].objectId;
				antArray[blockIdx.x].objectId = -1;
			}
			else
			{
				move(clocks, locks, antGrid, antArray, dim);
			}
		}
		else
		{
			move(clocks, locks, antGrid, antArray, dim);
		}
	}
}


__global__ void runDev(int* clocks, Lock* locks,int* antGrid, int* objectGrid, ant_t* antArray, object_t* objectArray, double* objectDist, int dim, int numAnts, int numObjects, int radius)
{	
	for(int i = 0; i < 1000; i++)
	{
		antRunDev(clocks, locks, antGrid, objectGrid, antArray, objectArray, objectDist, dim, numAnts, numObjects, radius);	
	}
}












